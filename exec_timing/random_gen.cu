#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void curand_call_kernel(hiprandState *state, int len){
    //threadIdx.x contains the index of the current thread within its block, 
    //and blockDim.x contains the number of threads in the block
    //and gridDim.x gives the number of blocks in a grid
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    float x;
    /* Generate pseudo-random uniforms */
    for(size_t i = index; i < len; i+=stride) {
        x = hiprand_uniform(&localState);
    }
    /* Copy state back to global memory */
    state[id] = localState;
}

float find_mean(float* arr, int len){
    float sum=0.0;
    for (int i = 0; i < len; ++i) sum += arr[i];
    return sum / len;
}

void time_curand_calls(hiprandState *devStates, int num_elem){
    const unsigned int threadsPerBlock = 256;
    const unsigned int blockCount = 64;
    float curand_times[20];
    for (int i = 0; i < 20; ++i){
        // Create the timer
        hipEvent_t total_start, total_stop;
        hipEventCreate(&total_start);
        hipEventCreate(&total_stop);

        // Start the timer
        hipEventRecord(total_start, 0);

        curand_call_kernel<<<blockCount, threadsPerBlock>>>(devStates, num_elem);

        // Stop the timer
        hipEventRecord(total_stop, 0);
        hipEventSynchronize(total_stop);
        std::cout << curand_times+i << std::endl;
        hipEventElapsedTime(curand_times+i, total_start, total_stop);
        std::cout << "Time to uniformly generate " << num_elem << " random numbers: " << curand_times[i] << " milliseconds" << std::endl;
    }
    std::cout << find_mean(curand_times, 20);
}

int main(void){

    float times[20];

    // start initializing the random number generator states
    const unsigned int threadsPerBlock = 256;
    const unsigned int blockCount = 64;
    const unsigned int totalThreads = threadsPerBlock * blockCount;

    for (int i = 0; i < 20; ++i){
        // Create the timer
        hipEvent_t total_start, total_stop;
        hipEventCreate(&total_start);
        hipEventCreate(&total_stop);

        // Start the timer for initializing the random number generator states
        hipEventRecord(total_start, 0);
        hiprandState *devStates;

        /* Allocate space for prng states on device */
        hipMalloc((void**)&devStates, totalThreads * sizeof(hiprandState));
    
        /* Setup prng states */
        setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);

        // Stop the timer
        hipEventRecord(total_stop, 0);
        hipEventSynchronize(total_stop);
        hipEventElapsedTime(times + i, total_start, total_stop);
        std::cout << "Time to initialize " << totalThreads << " thread prng states: " << times[i] << " milliseconds" << std::endl;
        /* Cleanup */
        hipFree(devStates);
        hipEventDestroy(total_start);
        hipEventDestroy(total_stop);
    }

    std::cout << find_mean(times, 20) << std::endl;

    hiprandState *devStates;
    /* Allocate space for prng states on device */
    hipMalloc((void**)&devStates, totalThreads * sizeof(hiprandState));
    // start the timer for making curand calls
    time_curand_calls(devStates, 10);
    hipFree(devStates);
}
