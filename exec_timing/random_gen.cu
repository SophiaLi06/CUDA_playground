#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void curand_call_kernel(hiprandState *state, int len){
    //threadIdx.x contains the index of the current thread within its block, 
    //and blockDim.x contains the number of threads in the block
    //and gridDim.x gives the number of blocks in a grid
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    float x;
    /* Generate pseudo-random uniforms */
    for(size_t i = index; i < len; i+=stride) {
        x = hiprand_uniform(&localState);
    }
    /* Copy state back to global memory */
    state[id] = localState;
}

void time_curand_calls(hiprandState *devStates, int num_elem){
    // Create the timer
    hipEvent_t total_start, total_stop;
    hipEventCreate(&total_start);
    hipEventCreate(&total_stop);

    // Start the timer
    hipEventRecord(total_start, 0);

    const unsigned int threadsPerBlock = 256;
    const unsigned int blockCount = 64;

    curand_call_kernel<<<blockCount, threadsPerBlock>>>(devStates, num_elem);

    // Stop the timer
    hipEventRecord(total_stop, 0);
    hipEventSynchronize(total_stop);
    float curand_time;
    hipEventElapsedTime(&curand_time, total_start, total_stop);
    std::cout << "Time to uniformly generate " << num_elem << " random numbers: " << curand_time << " milliseconds" << std::endl;
}

int main(void){
    // Create the timer
    hipEvent_t total_start, total_stop;
    hipEventCreate(&total_start);
    hipEventCreate(&total_stop);

    // Start the timer for initializing the random number generator states
    hipEventRecord(total_start, 0);

    // start initializing the random number generator states
    const unsigned int threadsPerBlock = 256;
    const unsigned int blockCount = 64;
    const unsigned int totalThreads = threadsPerBlock * blockCount;
    hiprandState *devStates;

    /* Allocate space for prng states on device */
    hipMalloc((void**)&devStates, totalThreads * sizeof(hiprandState));
    
    /* Setup prng states */
    setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);

    // Stop the timer
    hipEventRecord(total_stop, 0);
    hipEventSynchronize(total_stop);
    float setup_prng_time;
    hipEventElapsedTime(&setup_prng_time, total_start, total_stop);
    std::cout << "Time to initialize " << totalThreads << " thread prng states: " << setup_prng_time << " milliseconds" << std::endl;

    // start the timer for making hiprand calls
    time_curand_calls(devStates, 10)

}
