#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

int main(void){
    // Create the timer
    hipEvent_t total_start, total_stop;
    hipEventCreate(&total_start);
    hipEventCreate(&total_stop);

    // Start the timer
    hipEventRecord(total_start, 0);

    // start initializing the random number generator states
    const unsigned int threadsPerBlock = 512;
    const unsigned int blockCount = 64;
    const unsigned int totalThreads = threadsPerBlock * blockCount;
    hiprandState *devStates;

    /* Allocate space for prng states on device */
    hipMalloc((void**)&devStates, totalThreads * sizeof(hiprandState));
    
    /* Setup prng states */
    setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);

    // Stop the timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float setup_prng_time;
    hipEventElapsedTime(&setup_prng_time, start, stop);
    std::cout << "Time to initialize " << totalThreads << " thread prng states: " << setup_prng_time << std::endl;

}
